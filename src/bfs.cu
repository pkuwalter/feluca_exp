#include "hip/hip_runtime.h"
//Todo:time record on GPU


//#define PRINT_CHECK
//header file of memset()
#include<string.h>
#include<malloc.h>
#include<stdio.h>
#include<omp.h>

#include "graph.h"
#include "timer.h"
//#include "algorithm.h"
#include "hip/hip_runtime.h"

// The number of partitioning the duplicate chunk must be greater or equal to 1
#define ITERATE_IN_DUPLICATE 1
#define NUM_THREADS 1

#ifdef __CUDA_RUNTIME_H__
#define HANDLE_ERROR(err) if (err != hipSuccess) {	\
	printf("CUDA Error in %s at line %d: %s\n", \
			__FILE__, __LINE__, hipGetErrorString(hipGetLastError()));\
	exit(1);\
}
#endif  // #ifdef __CUDA_RUNTIME_H__    


void bfs_cpu(Graph_cpu *g,int *value_cpu,DataSize *dsize,int first_vertex)
{
	printf("BFS is running on CPU...............\n");
	timer_start();
	int vertex_num=dsize->vertex_num;
	int edge_num=dsize->edge_num;
	int edge_src,edge_dst;
	int *queue=(int *)malloc(sizeof(int)*vertex_num);
	memset(value_cpu,0,vertex_num*sizeof(int));
	value_cpu[first_vertex]=1;
	if(queue==NULL)
	{
		perror("Out of memory");
		exit(1);
	}

	int step=1;
	int incount=0;
	int outcount=0;
	queue[incount++]=first_vertex;

	while(incount > outcount)
	{
		int vertex_id=queue[outcount++];
		for (int i = g->vertex_begin[vertex_id]; i < g->vertex_begin[vertex_id+1]; ++i)
		{
			int dst_id=g->vertex_dst[i];
			step=value_cpu[vertex_id];
			if (value_cpu[dst_id]==0)
			{
				value_cpu[dst_id]=step+1;
				queue[incount++]=dst_id;
			}
		}

#ifdef PRINT_CHECK
		printf("\n");
		for (int i = 0; i < 15 && i<vertex_num+1; ++i)
		{
			printf("%d\t", value_cpu[i]);
		}
		printf("\n");
#endif

	}
	double total_time=timer_stop();
	printf("Total time of bfs_cpu is %.3fms\n",total_time);
}

// print info about bfs values
void print_bfs_values(const int * const values, int const size) {
	int visited = 0;
	int step = 0;
	int first = 0;

	// get the max step and count the visited
	for (int i = 0; i < size; i++) {
		if (values[i] != 0) {
			visited++;
			if (values[i] > step) step = values[i];
			if (values[i] == 1) first = i;
		}
	}
	// count vertices of each step
	if (step == 0) return;
	int * m = (int *) malloc((step + 1)*sizeof(int));
	memset(m,0,sizeof(int)*(step+1));
	for (int i = 0; i < size; i++) {
		m[values[i]]++;
	}
	// print result info
	printf("\tSource = %d, Step = %d, Visited = %d\n", first, step, visited);
	printf("\tstep\tvisit\n");
	for (int i = 1; i <= step; i++) {
		printf("\t%d\t%d\n", i, m[i]);
	}
	printf("\n");
	free(m);
}

static __global__ void  bfs_kernel_duplicate(  
		const int edge_num,
		const int * const edge_src,
		const int * const edge_dest,
		int * const values,
		const int step)
{
	// total thread number & thread index of this thread
	int n = blockDim.x * gridDim.x;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// step counter
	int curStep = step;
	int nextStep = curStep + 1;
	// proceeding loop
	for (int i = index; i < edge_num; i +=n) {		
		if (values[edge_src[i]] == curStep && values[edge_dest[i]] == 0) {
			values[edge_dest[i]] = nextStep;
		}
	}
}
static __global__ void bfs_kernel_local(  
		const int edge_num,
		const int * const edge_src,
		const int * const edge_dest,
		int * const values,
		const int step,
		int * const continue_flag)
{

	// total thread number & thread index of this thread
	int n = blockDim.x * gridDim.x;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	// continue flag for each thread
	int flag = 0;
	int curStep = step;
	int nextStep = curStep + 1;

	for (int i = index; i < edge_num; i +=n) {		
		if(values[edge_src[i]]==curStep && values[edge_dest[i]]==0)	
		{
			values[edge_dest[i]]=nextStep;
			flag = 1;
		}	
	}
	// update flag
	if (flag == 1) *continue_flag = 1;
}
static __global__ void kernel_make_bitmap(
		int const vertex_num,
		int const * const values,
		int * const bitmap,
		int const targe)
{
	int const n=blockDim.x*gridDim.x;
	int const tid=threadIdx.x+blockDim.x*blockIdx.x;
	for (int i = tid; i < vertex_num; i+=n)
	{
		int const v=__ballot(values[i]==targe);
		bitmap[i>>5]=v;
	}
}

static __global__ void  kernel_extract_bitmap(  
		int const vertex_num,
		int const * const bitmap,
		int * const values,
		int const targe)
{
	int const n=blockDim.x*gridDim.x;
	int const tid=threadIdx.x+blockIdx.x*blockDim.x;
	int const mask=1<<(tid & 31);
	for (int i = tid; i < vertex_num; i=i+n)
	{
		if(bitmap[i>>5]&mask) values[i]=targe;
	}
}


void merge_bitmap_on_cpu(
		int const bitmap_len,
		int const gpu_num,
		int * const *  bitmap,
		int * const  buffer,
		int &flag)
{
	int i,id;
	omp_set_num_threads(NUM_THREADS);	
#pragma omp parallel private(i)
	{
		id=omp_get_thread_num(); 
		for (i = id; i < bitmap_len; i=i+NUM_THREADS)
		{
			buffer[i]=0;
			int t=0;
			for (int j = 0; j < gpu_num; ++j)
			{
				t=t| bitmap[j][i];
				if(t) 
				{
					buffer[i]=t;
					flag=1;
					break;
				}
			}
		}

	}
}

void merge_bitmap_on_cpu_1(
		int const bitmap_len,
		int const duplicate_num,
		int const gpu_num,
		int * const *  bitmap,
		int * const  buffer,
		int &flag)
{
	int i,id;
	{
		for (i = 0; i < bitmap_len*duplicate_num; i++)
		{
			int t=0;
			for (int j = 0; j < gpu_num; ++j)
			{
				t=t| bitmap[j][i];

			}
			bitmap[0][i]=t;
		}
		for(i=0; i< bitmap_len;i++)
		{
			int t=0;
			buffer[i]=0;
			for(int j=0; j<duplicate_num;j++)
			{
				int *p=bitmap[0]+bitmap_len*j;
				t=t|p[i];
				if(t)
				{
					buffer[i]=t;
flag=1;
					break;
				}
			}

		}

	}
}

void Gather_result(
		int vertex_num,
		int gpu_num,
		int * const * const h_value,
		int * const value_gpu)
{
	omp_set_num_threads(NUM_THREADS);
	int j,id;	
#pragma omp parallel private(j)
	for (int i = 0; i < gpu_num; ++i)
	{
		//		int *edge_dest=g[i]->edge_local_dst;
		//	int size=g[i]->edge_num-g[i]->edge_duplicate_num;
		id=omp_get_thread_num(); 
		for (j = id; j <vertex_num	; j=j+NUM_THREADS)
		{
			if(h_value[i][j]>0)
				value_gpu[j]=h_value[i][j];
		}
	}
}
/* BFS algorithm on GPU */
void bfs_gpu(Graph **g,int gpu_num,int *value_gpu,DataSize *dsize, int first_vertex, int *copy_num, int **position_id)
{
	printf("BFS is running on GPU...............\n");
	printf("Start malloc edgelist...\n");
	/* TODO : edgelsit store twices */
	/* Inite value*/
	value_gpu[first_vertex]=1;
	// TODO : can be deleted
	int **h_value=(int **)malloc(sizeof(int *)* gpu_num);
	int **h_flag=(int **)malloc(sizeof(int *)*gpu_num);
	int vertex_num=dsize->vertex_num;
	int **d_edge_local_src=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_edge_local_dst=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_edge_duplicate_src=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_edge_duplicate_dst=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_value=(int **)malloc(sizeof(int *)*gpu_num);
	int **d_flag=(int **)malloc(sizeof(int *)*gpu_num);

	//add
	int  bitmap_len=(vertex_num+sizeof(int)*8-1)/(sizeof(int)*8);
	int  **h_bitmap=(int **)malloc(sizeof(int *)*gpu_num); 
	int  **d_bitmap=(int **)malloc(sizeof(int *)*gpu_num); 
	int *buff_bitmap=(int *)malloc(sizeof(int)*bitmap_len);


	/* determine the size of duplicate vertex in one process*/
	int tmp_per_size = min_num_duplicate_edge(g,gpu_num);
	int duplicate_per_size=tmp_per_size/ITERATE_IN_DUPLICATE;
	int iterate_in_duplicate=ITERATE_IN_DUPLICATE+1;
	int *last_duplicate_per_size=(int *)malloc(sizeof(int)*gpu_num);
	memset(last_duplicate_per_size,0,sizeof(int)*gpu_num);



	for (int i = 0; i < gpu_num; ++i)
	{
		h_value[i]=(int *)malloc(sizeof(int)*(vertex_num+1));
		memset(h_value[i],0,sizeof(int)*(vertex_num+1));
		h_value[i][first_vertex]=1;
		h_flag[i]=(int *)malloc(sizeof(int));

		//add
		h_bitmap[i]=(int *)malloc(sizeof(int)*(bitmap_len*iterate_in_duplicate));
		memset(h_bitmap[i],0,sizeof(int)*(bitmap_len*iterate_in_duplicate));
	}



	/*Cuda Malloc*/
	/* Malloc stream*/
	hipStream_t **stream;
	hipEvent_t tmp_start,tmp_stop;
	stream=(hipStream_t **)malloc(gpu_num*sizeof(hipStream_t*));

	hipEvent_t * start_duplicate,*stop_duplicate,*start_local,*stop_local,*start_asyn,*stop_asyn,*start,*stop;
	start_duplicate=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	stop_duplicate=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	start_local=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	stop_local=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	start_asyn=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	stop_asyn=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	start=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));
	stop=(hipEvent_t *)malloc(gpu_num*sizeof(hipEvent_t));

	for (int i = 0; i < gpu_num; ++i)
	{
		hipSetDevice(i);
		stream[i]=(hipStream_t *)malloc((iterate_in_duplicate+1)*sizeof(hipStream_t));
		HANDLE_ERROR(hipEventCreate(&start_duplicate[i],0));
		HANDLE_ERROR(hipEventCreate(&stop_duplicate[i],0));
		HANDLE_ERROR(hipEventCreate(&start_local[i],0));
		HANDLE_ERROR(hipEventCreate(&stop_local[i],0));  
		HANDLE_ERROR(hipEventCreate(&start_asyn[i],0));
		HANDLE_ERROR(hipEventCreate(&stop_asyn[i],0));
		HANDLE_ERROR(hipEventCreate(&start[i],0));
		HANDLE_ERROR(hipEventCreate(&stop[i],0));
		HANDLE_ERROR(hipEventCreate(&tmp_start,0));
		HANDLE_ERROR(hipEventCreate(&tmp_stop,0));

		for (int j = 0; j <= iterate_in_duplicate; ++j)
		{
			HANDLE_ERROR(hipStreamCreate(&stream[i][j]));
		}
	}


	for (int i = 0; i < gpu_num; ++i)
	{
		hipSetDevice(i);
		int out_size=g[i]->edge_duplicate_num;
		int local_size=g[i]->edge_num - out_size;

		HANDLE_ERROR(hipMalloc((void **)&d_edge_duplicate_src[i],sizeof(int)*out_size));
		HANDLE_ERROR(hipMalloc((void **)&d_edge_duplicate_dst[i],sizeof(int)*out_size));
		HANDLE_ERROR(hipMalloc((void **)&d_bitmap[i],sizeof(int)*(bitmap_len*iterate_in_duplicate)));

		if (duplicate_per_size!=0 && duplicate_per_size < out_size)
		{
			for (int j = 1; j < iterate_in_duplicate; ++j)
			{
				HANDLE_ERROR(hipMemcpyAsync((void *)(d_edge_duplicate_src[i]+(j-1)*duplicate_per_size),(void *)(g[i]->edge_duplicate_src+(j-1)*duplicate_per_size),sizeof(int)*duplicate_per_size,hipMemcpyHostToDevice, stream[i][j-1]));
				HANDLE_ERROR(hipMemcpyAsync((void *)(d_edge_duplicate_dst[i]+(j-1)*duplicate_per_size),(void *)(g[i]->edge_duplicate_dst+(j-1)*duplicate_per_size),sizeof(int)*duplicate_per_size,hipMemcpyHostToDevice, stream[i][j-1]));
				HANDLE_ERROR(hipMemcpyAsync((void *)(d_bitmap[i]+(j-1)*bitmap_len),(void *)(h_bitmap[i]+(j-1)*bitmap_len),sizeof(int)*(bitmap_len),hipMemcpyHostToDevice,stream[i][j-1]));
			}
		}

		last_duplicate_per_size[i]=g[i]->edge_duplicate_num-duplicate_per_size * (iterate_in_duplicate-1);           
		if (last_duplicate_per_size[i]>0 && iterate_in_duplicate>1 )
		{
			HANDLE_ERROR(hipMemcpyAsync((void *)(d_edge_duplicate_src[i]+(iterate_in_duplicate-1)*duplicate_per_size),(void *)(g[i]->edge_duplicate_src+(iterate_in_duplicate-1)*duplicate_per_size),sizeof(int)*last_duplicate_per_size[i],hipMemcpyHostToDevice, stream[i][iterate_in_duplicate-1]));
			HANDLE_ERROR(hipMemcpyAsync((void *)(d_edge_duplicate_dst[i]+(iterate_in_duplicate-1)*duplicate_per_size),(void *)(g[i]->edge_duplicate_dst+(iterate_in_duplicate-1)*duplicate_per_size),sizeof(int)*last_duplicate_per_size[i],hipMemcpyHostToDevice, stream[i][iterate_in_duplicate-1]));
			HANDLE_ERROR(hipMemcpyAsync((void *)(d_bitmap[i]+(iterate_in_duplicate-1)*bitmap_len),(void *)(h_bitmap[i]+(iterate_in_duplicate-1)*bitmap_len),sizeof(int)*bitmap_len,hipMemcpyHostToDevice,stream[i][iterate_in_duplicate-1]));
		}


		HANDLE_ERROR(hipMalloc((void **)&d_edge_local_src[i],sizeof(int)*local_size));
		HANDLE_ERROR(hipMalloc((void **)&d_edge_local_dst[i],sizeof(int)*local_size));
		HANDLE_ERROR(hipMemcpyAsync((void *)d_edge_local_src[i],(void *)g[i]->edge_local_src,sizeof(int)*local_size,hipMemcpyHostToDevice,stream[i][iterate_in_duplicate]));
		HANDLE_ERROR(hipMemcpyAsync((void *)d_edge_local_dst[i],(void *)g[i]->edge_local_dst,sizeof(int)*local_size,hipMemcpyHostToDevice,stream[i][iterate_in_duplicate]));

		HANDLE_ERROR(hipMalloc((void **)&d_value[i],sizeof(int)*(vertex_num+1)));
		HANDLE_ERROR(hipMemcpyAsync((void *)d_value[i],(void *)h_value[i],sizeof(int)*(vertex_num+1),hipMemcpyHostToDevice,stream[i][0]));

		HANDLE_ERROR(hipMalloc((void **)&d_flag[i],sizeof(int)));


	}
	printf("Malloc is finished!\n");

	/* Before While: Time Initialization */
	float *duplicate_compute_time,*local_compute_time,*compute_time,*total_compute_time,*extract_bitmap_time;
	float gather_time=0.0;
	float cpu_gather_time=0.0;
	float total_time=0.0;
	float record_time=0.0;
	duplicate_compute_time=(float *)malloc(sizeof(float)*gpu_num);
	local_compute_time=(float *)malloc(sizeof(float)*gpu_num);
	compute_time=(float *)malloc(sizeof(float)*gpu_num);
	total_compute_time=(float *)malloc(sizeof(float)*gpu_num);
	extract_bitmap_time=(float *)malloc(sizeof(float)*gpu_num);

	memset(duplicate_compute_time,0,sizeof(float)*gpu_num);
	memset(local_compute_time,0,sizeof(float)*gpu_num);
	memset(compute_time,0,sizeof(float)*gpu_num);


	/* Before While: Variable Initialization */
	int flag=0;
	int step=1;
	int local_edge_num=0;

#ifdef PRINT_CHECK
	for (int i = 0; i < gpu_num; ++i)
	{

		HANDLE_ERROR(hipMemcpy(h_value[i],d_value[i],sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost));
		printf("Before while --> check value\n");
		printf("value:\n");
		for (int j = 0; j < vertex_num+1; ++j)
		{
			printf("%d\t", h_value[i][j]);
		}
		printf("\nDuplicate_edgelist\n");
		for (int j = 0; j < g[i]->edge_duplicate_num ; ++j)
		{
			printf("( %d, %d )\t",g[i]->edge_duplicate_src[j],g[i]->edge_duplicate_dst[j]);
		}
		printf("\nlocal_edgelist\n");
		for (int j = 0; j < g[i]->edge_num- g[i]->edge_duplicate_num; ++j)
		{
			printf("( %d, %d )\t",g[i]->edge_local_src[j],g[i]->edge_local_dst[j]);
		}
		printf("\n");
	}
#endif

	/* one iteration */
	printf("Computing......\n");
	do
	{
		flag=0;
		for (int i = 0; i <gpu_num; ++i)
		{		
			memset(h_flag[i],0,sizeof(int));
			hipSetDevice(i);
            HANDLE_ERROR(hipMemset(d_bitmap[i],0,sizeof(int)*(bitmap_len*iterate_in_duplicate)));	
			HANDLE_ERROR(hipMemset(d_flag[i],0,sizeof(int)));

			HANDLE_ERROR(hipEventRecord(start_duplicate[i], stream[i][0]));
			//kernel of duplicate edgelist
			if (duplicate_per_size!=0 && duplicate_per_size < g[i]->edge_duplicate_num)
			{
				for (int j = 1; j < iterate_in_duplicate; ++j)
				{				
					bfs_kernel_duplicate<<<208,128,0,stream[i][j-1]>>>(
							duplicate_per_size,
							d_edge_duplicate_src[i]+(j-1)*duplicate_per_size,
							d_edge_duplicate_dst[i]+(j-1)*duplicate_per_size,
							d_value[i],
							step);
					kernel_make_bitmap<<<208,128,0,stream[i][j-1]>>>(
							vertex_num,
							d_value[i],
							d_bitmap[i]+(j-1)*bitmap_len,
							(step+1));
					//HANDLE_ERROR(hipMemcpyAsync((void *)(h_bitmap[i]+(j-1)*bitmap_len),(void *)(d_bitmap[i]+(j-1)*bitmap_len),sizeof(int)*(bitmap_len),hipMemcpyDeviceToHost,stream[i][j-1]));
					//HANDLE_ERROR(hipMemcpy((void *)(h_bitmap[i]+(j-1)*bitmap_len),(void *)(d_bitmap[i]+(j-1)*bitmap_len),sizeof(int)*(bitmap_len),hipMemcpyDeviceToHost,stream[i][j-1]));
				}
			}

			last_duplicate_per_size[i]=g[i]->edge_duplicate_num-duplicate_per_size * (iterate_in_duplicate-1);           
			if (last_duplicate_per_size[i]>0 && iterate_in_duplicate>1  )
			{
				// The size of edge list in last block is different in every gpu
				bfs_kernel_duplicate<<<208,128,0,stream[i][iterate_in_duplicate-1]>>>(
						last_duplicate_per_size[i],
						d_edge_duplicate_src[i]+(iterate_in_duplicate-1)*duplicate_per_size,
						d_edge_duplicate_dst[i]+(iterate_in_duplicate-1)*duplicate_per_size,
						d_value[i],
						step);
				kernel_make_bitmap<<<208,128,0,stream[i][iterate_in_duplicate-1]>>>(
						vertex_num,
						d_value[i],
						d_bitmap[i]+(iterate_in_duplicate-1)*bitmap_len,
						step+1
						);
				//HANDLE_ERROR(hipMemcpyAsync((void *)(h_bitmap[i]+(iterate_in_duplicate-1)*bitmap_len),(void *)(d_bitmap[i]+(iterate_in_duplicate-1)*bitmap_len),sizeof(int)*(bitmap_len),hipMemcpyDeviceToHost,stream[i][iterate_in_duplicate-1]));
			   //HANDLE_ERROR(hipMemcpy((void *)(h_bitmap[i]+(iterate_in_duplicate-1)*bitmap_len),(void *)(d_bitmap[i]+(iterate_in_duplicate-1)*bitmap_len),sizeof(int)*(bitmap_len),hipMemcpyDeviceToHost));
			}
			HANDLE_ERROR(hipEventRecord(stop_duplicate[i], stream[i][iterate_in_duplicate-1]));
            
           HANDLE_ERROR(hipMemcpy(h_bitmap[i],d_bitmap[i],sizeof(int)*(bitmap_len*iterate_in_duplicate),hipMemcpyDeviceToHost));			
#ifdef PRINT_CHECK_1
			printf("The value after bfs_duplicate_kernel\n");
			HANDLE_ERROR(hipMemcpy(h_value[i],d_value[i],sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipMemcpy(h_bitmap[i],d_bitmap[i],sizeof(int)*(bitmap_len*iterate_in_duplicate),hipMemcpyDeviceToHost));
			printf("@@value\n");
			for (int j = 0; j < vertex_num+1 && j<10; ++j)
			{
				printf("%d\t", h_value[i][j]);
			}
			printf("\n@@bitmap:\n");
			for(int j=0;j<bitmap_len*iterate_in_duplicate;j++)
			{
				printf("%d\t",h_bitmap[i][j]);
			}
			printf("\n\n");
#endif

			HANDLE_ERROR(hipEventRecord(start_local[i], stream[i][iterate_in_duplicate]));
			//local+flag
			local_edge_num=g[i]->edge_num-g[i]->edge_duplicate_num;
			if (local_edge_num>0)
			{
				bfs_kernel_local<<<208,128,0,stream[i][iterate_in_duplicate]>>>(
						local_edge_num,
						d_edge_local_src[i],
						d_edge_local_dst[i],
						d_value[i],
						step,
						d_flag[i]);			
				HANDLE_ERROR(hipMemcpyAsync(h_flag[i], d_flag[i],sizeof(int),hipMemcpyDeviceToHost,stream[i][iterate_in_duplicate]));	    
			}
			HANDLE_ERROR(hipEventRecord(stop_local[i],stream[i][iterate_in_duplicate]));


#ifdef PRINT_CHECK_1
			printf("The value after bfs_local_kernel\n");
			HANDLE_ERROR(hipMemcpy(h_value[i],d_value[i],sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipMemcpy(h_bitmap[i],d_bitmap[i],sizeof(int)*(bitmap_len*iterate_in_duplicate),hipMemcpyDeviceToHost));
			HANDLE_ERROR(hipMemcpy(h_flag[i], d_flag[i],sizeof(int),hipMemcpyDeviceToHost));
			printf("@@ value\n");
			for (int j = 0; j < vertex_num+1 && j<10; ++j)
			{
				printf("%d\t", h_value[i][j]);
			}
			printf("\n @@GPU flag:%d\n",h_flag[i][0]);
			printf("bitmap:\n");
			for(int j=0;j<bitmap_len*iterate_in_duplicate;j++)
			{
				printf("%d\t",h_bitmap[i][j]);
			}
			printf("\n\n");
#endif

		}


		//merge bitmap on gpu
		double t1=omp_get_wtime();
		merge_bitmap_on_cpu_1(bitmap_len, iterate_in_duplicate, gpu_num, h_bitmap, buff_bitmap,flag);
		double t2=omp_get_wtime();
		record_time=(t2-t1)*1000;
		gather_time+=record_time;


#ifdef PRINT_CHECK_1
		printf("-----------------------------After merge\n");
		printf("value:\n");
		for (int i = 0; i < gpu_num; ++i)
		{
			HANDLE_ERROR(hipMemcpy(h_value[i],d_value[i],sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost));
			for (int j = 0; j < vertex_num+1 && j<10; ++j)
			{
				printf("%d\t", h_value[i][j]);
			}
			printf("\n");

		}
        
		printf("@@bitmap:\n");
	for (int i = 0; i < bitmap_len; ++i)
	{
		printf("%d\n",buff_bitmap[i]);
	}
		printf("@@ flag %d\n\n", flag);


#endif


		for (int i = 0; i < gpu_num; ++i)
		{
			hipSetDevice(i);
			//extract bitmap to the value
			HANDLE_ERROR(hipMemcpyAsync(d_bitmap[i], buff_bitmap,sizeof(int)*bitmap_len,hipMemcpyHostToDevice,stream[i][0]));
			HANDLE_ERROR(hipEventRecord(start_asyn[i], stream[i][0]));
			kernel_extract_bitmap<<<256,108,0,stream[i][0]>>>
				(  
				 vertex_num,
				 d_bitmap[i],
				 d_value[i],
				 step+1
				);		
			HANDLE_ERROR(hipEventRecord(stop_asyn[i], stream[i][0]));
			HANDLE_ERROR(hipMemset(d_bitmap[i],0,sizeof(int)*(bitmap_len*iterate_in_duplicate)));	
		}

#ifdef PRINT_CHECK_1
		//HANDLE_ERROR(hipMemcpy(h_flag[i], d_flag[i],sizeof(int),hipMemcpyDeviceToHost));
		printf("-----------------------------After extract\n");
		printf("value\n");
		for (int i = 0; i < gpu_num; ++i)
		{
			HANDLE_ERROR(hipMemcpy(h_value[i],d_value[i],sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost));
			for (int j = 0; j < vertex_num+1 && j<10; ++j)
			{
				printf("%d\t", h_value[i][j]);

			}
			printf("\n\n");
		}


#endif


#ifdef PRINT_CHECK_1
		printf("The value after bfs_extral_bitmap/   before next iteration\n");
		printf("buff_bitmap\n");
		for(int j=0 ; j<bitmap_len;j++)
			printf("%d\t",buff_bitmap[j]);
		printf("\nvalue\n");
		for(int i=0;i<gpu_num;i++)
		{
			HANDLE_ERROR(hipMemcpy(h_value[i],d_value[i],sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost));
			for (int j = 0; j < vertex_num+1 && j<10; ++j)
			{
				printf("%d\t", h_value[i][j]);
			}
			printf("\n");
		}
		printf("\nCPU flag:%d\n",flag);
#endif


		for (int i = 0; i < gpu_num; ++i)
		{
			flag=flag||h_flag[i][0];
		}
		step++;

		//collect time  different stream
		for (int i = 0; i < gpu_num; ++i)
		{
			hipSetDevice(i);
			HANDLE_ERROR(hipEventSynchronize(stop_duplicate[i]));
			HANDLE_ERROR(hipEventSynchronize(stop_local[i]));
			HANDLE_ERROR(hipEventSynchronize(stop_asyn[i]));

			HANDLE_ERROR(hipEventElapsedTime(&record_time, start_duplicate[i], stop_duplicate[i]));
			duplicate_compute_time[i]+=record_time;
			HANDLE_ERROR(hipEventElapsedTime(&record_time, start_local[i], stop_local[i]));  
			local_compute_time[i]+=record_time;
			HANDLE_ERROR(hipEventElapsedTime(&record_time, start_asyn[i], stop_asyn[i]));  
			extract_bitmap_time[i]+=record_time;
			total_compute_time[i]=duplicate_compute_time[i]+extract_bitmap_time[i]-local_compute_time[i]>0?(duplicate_compute_time[i]+extract_bitmap_time[i]):local_compute_time[i];
		}		
	}while(flag && step<1000);


	//Todo to get the true value of local vertice and duplicate vertice
	for (int i = 0; i < gpu_num; ++i)
	{
		hipSetDevice(i);
		hipMemcpyAsync((void *)h_value[i],(void *)d_value[i],sizeof(int)*(vertex_num+1),hipMemcpyDeviceToHost,stream[i][0]);
	}

	printf("Gather result on cpu....\n");
	Gather_result(vertex_num+1,gpu_num,h_value,value_gpu);

	printf("Time print\n");

	//collect the information of time 
	float total_time_n=0.0;
	for (int i = 0; i < gpu_num; ++i)
	{
		if(total_time_n<total_compute_time[i])
			total_time_n=total_compute_time[i];
	}
	total_time=total_time_n>gather_time?total_time_n:gather_time;

//	printf("Total time of bfs_gpu is %.3f ms\n",total_time);
	printf("Elapsed time of bfs_gpu is %.3f ms\n", total_time/step);
	printf("%d step\n",step);
	printf("-------------------------------------------------------\n");
	printf("Detail:\n");
	printf("\n");
	for (int i = 0; i < gpu_num; ++i)
	{
		printf("GPU %d\n",i);
		printf("Duplicate_Compute_Time(include pre-stage):  %.3f ms\n", duplicate_compute_time[i]/step);
		printf("local_Compute_Time:                     %.3f ms\n", local_compute_time[i]/step);
		printf("Total Compute_Time                      %.3f ms\n", total_compute_time[i]/step);
		printf("Extract_Bitmap_Time                     %.3f ms\n", extract_bitmap_time[i]/step);
	}
	printf("CPU \n");
	printf("CPU_Gather_Time:                            %.3f ms\n", gather_time/step);
	printf("--------------------------------------------------------\n");

	//clean

	for (int i = 0; i < gpu_num; ++i)
	{
		hipSetDevice(i);
		HANDLE_ERROR(hipEventDestroy(start[i]));
		HANDLE_ERROR(hipEventDestroy(stop[i]));
		HANDLE_ERROR(hipFree(d_edge_duplicate_src[i]));
		HANDLE_ERROR(hipFree(d_edge_duplicate_dst[i]));
		HANDLE_ERROR(hipFree(d_edge_local_src[i]));
		HANDLE_ERROR(hipFree(d_edge_local_dst[i]));
		HANDLE_ERROR(hipFree(d_value[i]));
		HANDLE_ERROR(hipFree(d_flag[i]));

		HANDLE_ERROR(hipDeviceReset());
		//error 
		//free(h_value[i]);
		free(h_flag[i]);
		free(stream[i]);
	}

	free(duplicate_compute_time);
	free(local_compute_time);
	free(compute_time);
}